// Name:Kyla 
// Vector Dot product on 1 block 
// nvcc HW8.cu -o temp
/*
 What to do:
 This code uses the CPU to compute the dot product of two vectors of length N. 
 It includes a skeleton for setting up a GPU dot product, but that part is currently empty.
 Additionally, the CPU code is somewhat convoluted, but it is structured this way to parallel 
 the GPU code you will need to write. The program will also verify whether you have correctly 
 implemented the dot product on the GPU.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

// Defines
#define N 823 // Length of the vector

// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
float DotCPU, DotGPU;
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid
float Tolerance = 0.01;

// Function prototypes
void cudaErrorCheck(const char *, int);
void setUpDevices();
void allocateMemory();
void innitialize();
void dotProductCPU(float*, float*, int);
__global__ void dotProductGPU(float*, float*, float*, int);
bool  check(float, float, float);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
	BlockSize.x = 1000;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = 1; //(int)(((N-1)/BlockSize.x)+1);
	GridSize.y = 1;
	GridSize.z = 1;
}

// Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
	
	// Device "GPU" Memory
	hipMalloc(&A_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
}

// Loading values into the vectors that we will add.
void innitialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(2*i);
	}
}

// Adding vectors a and b on the CPU then stores result in vector c.
void dotProductCPU(float *a, float *b, float *C_CPU, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		C_CPU[id] = a[id] * b[id];
	}
	
	for(int id = 1; id < n; id++)
	{ 
		C_CPU[0] += C_CPU[id];
	}
}

// This is the kernel. It is the function that will run on the GPU.
// It adds vectors a and b on the GPU then stores result in vector c.
__global__ void dotProductGPU(float *a, float *b, float *C_GPU, int n)
{
	int id = threadIdx.x;//+blockIdx.x*blockDim.x, only using one block in this code

	if(id<n)//makes sure we are in our yard
	{ 
		C_GPU[id] = a[id] * b[id];//multiplies elements
	}

	 __syncthreads();//make sure everyone is done before we start adding things

	 if(id<n)
		{//makes sure we are in our yard
			int size=n;//how many elements we need to add. 

			while(size>1)//while we have more than one thing to add. once its one we are done. hey that rhymed 
			{

				if(size%2==0)//checking if the number of things were adding is even,if yes it's easy. 
				{
					if(id<size/2)// if the id is smaller than size/2, it's gonna be put to work
						{
							C_GPU[id]=C_GPU[id]+C_GPU[id+size/2];// adds current id to id+size/2. like adding 0 and 500, in a size of 1000
						}

				}
				else{// if not even...
					if(id==0)//if its id is 0, then we want to add the last value to the first, to make it even
						{
						C_GPU[id] = C_GPU[id]+C_GPU[id+size-1];
						}
				size=size-1;// take away one from size, since we added it to id 0

					if(id<size/2)// now just do the same thing we did for even
						{
						C_GPU[id]=C_GPU[id]+C_GPU[id+size/2];
						}
			}

			size=size/2;// cut in half
			__syncthreads();// make sure everyone is on the same page
			}
		
		}
	 }



// Checking to see if anything went wrong in the vector addition.
bool check(float cpuAnswer, float gpuAnswer, float tolerence)
{
	double percentError;
	
	percentError = abs((gpuAnswer - cpuAnswer)/(cpuAnswer))*100.0;
	printf("\n\n percent error = %lf\n", percentError);
	
	if(percentError < Tolerance) 
	{
		return(true);
	}
	else 
	{
		return(false);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

// Cleaning up memory after we are finished.
void CleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU); 
	free(B_CPU); 
	free(C_CPU);
	
	hipFree(A_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
}

int main()
{
	timeval start, end;
	long timeCPU, timeGPU;
	
	// Setting up the GPU
	setUpDevices();
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	innitialize();
	
	// Adding on the CPU
	gettimeofday(&start, NULL);
	dotProductCPU(A_CPU, B_CPU, C_CPU, N);
	DotCPU = C_CPU[0];
	gettimeofday(&end, NULL);
	timeCPU = elaspedTime(start, end);
	
	// Adding on the GPU
	gettimeofday(&start, NULL);
	
	// Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	dotProductGPU<<<GridSize,BlockSize>>>(A_GPU, B_GPU, C_GPU, N);
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	DotGPU = C_CPU[0]; // C_GPU was copied into C_CPU.
	
	// Making sure the GPU and CPU wiat until each other are at the same place.
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);

	gettimeofday(&end, NULL);
	timeGPU = elaspedTime(start, end);
	
	// Checking to see if all went correctly.
	if(check(DotCPU, DotGPU, Tolerance) == false)
	{
		printf("\n\n Something went wrong in the GPU dot product.\n");
	}
	else
	{
		printf("\n\n You did a dot product correctly on the GPU");
		printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
		printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
	}
	
	// Your done so cleanup your room.	
	CleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n\n");
	
	return(0);
}


